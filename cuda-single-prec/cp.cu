#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include "math.h"

static inline void check(hipError_t err, const char* context) {
    if(err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1) / b;
}
static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

__global__ void normalization_kernel(int ny, int nx, float* data, float* ntdata) {
    int y = blockIdx.x;
    if(y >= ny) return;

    float s = 0.0;
    for(int x=0; x < nx; x++) {
        float v = data[x+y*nx];
        s += v;
    }
    float m = s / (float) nx;
    float rs = 0.0;

    for(int x=0; x < nx; x++) {
        float v = data[x + y * nx];
        rs += ((v - m) * (v - m));
    }
    float r = sqrt(rs);
    for(int x=0; x < nx; x++) {
        float v = ((data[x + y * nx]) - m ) / r;
        ntdata[y + x * ny] = v;
    }
}

__global__ void matmul_kernel(int ny, int nx, float* ntdata, float* r){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if(i >= ny || j >= ny) return;
    float s = 0.0;
    if(i <= j) {
        for(int k=0; k < nx; k++) {
            float x = ntdata[ny * k + j];
            float y = ntdata[ny * k + i];
            s += (x * y);
        }
    }
    r[j + i * ny] = s;
}

void correlate(int ny, int nx, const float* data, float* result) {
    int n = ny * nx * sizeof(float);
    int rn = ny * ny * sizeof(float);

    float* dGPU = NULL;
    CHECK(hipMalloc((void**)&dGPU, n));

    float* ntGPU = NULL;
    CHECK(hipMalloc((void**)&ntGPU, n));

    float* rGPU = NULL;
    CHECK(hipMalloc((void**)&rGPU, rn));

    CHECK(hipMemcpy(dGPU, data, n, hipMemcpyHostToDevice));

    int nBlocks = roundup(ny, 64);
    {
        normalization_kernel<<<nBlocks, 1>>>(ny, nx, dGPU, ntGPU);
        CHECK(hipGetLastError());
    }
    {
        dim3 dimBlock(16, 16);
        dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
        matmul_kernel<<<dimGrid, dimBlock>>>(ny, nx, ntGPU, rGPU);
        CHECK(hipGetLastError());
    }
    CHECK(hipMemcpy(result, rGPU, rn, hipMemcpyDeviceToHost));
    CHECK(hipFree(dGPU));
    CHECK(hipFree(ntGPU));
    CHECK(hipFree(rGPU));
}
